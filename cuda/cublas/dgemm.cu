
#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <stdio.h>

const int N = 4;

int main(){
  double *x,*y,*c;

  hipblasHandle_t handle;
  hipblasCreate(&handle); 

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*N*sizeof(double));
  hipMallocManaged(&y, N*N*sizeof(double));
  hipMallocManaged(&c, N*N*sizeof(double));

  for(int col=0;col<N;col++){
    for(int row=0;row<N;row++){
      x[col*N + row ] = (col+1)*2 + (row+1);
      y[col*N + row ] = (row+1)*3 - (col+1);
    }
  }


  for(int col=0;col<N;col++){
    for(int row=0;row<N;row++)
      printf("%lf ",x[row*N + col]);
    printf("\n");
  }

    printf("\n");
  for(int col=0;col<N;col++){
    for(int row=0;row<N;row++)
      printf("%lf ",y[row*N + col]);
    printf("\n");
  }

	hipblasOperation_t transa,transb;


	transa = HIPBLAS_OP_N;
	transb = HIPBLAS_OP_N;	

  double alpha = 1;

hipblasDgemm(handle,transa,transb,N,N,N,&alpha,x,N,y,N,&alpha,c,N);


  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

    printf("\n");
  for(int col=0;col<N;col++){
    for(int row=0;row<N;row++)
      printf("%lf ",c[row*N + col]);
    printf("\n");
  }


  // Free memory
  hipFree(x);
  hipFree(y);
  hipFree(c);


  // Undefined Error.. Why? 
  //cublasDestory(handle);

  return 0;
}
